#include "hip/hip_runtime.h"
#include "codelets.cuh"
#include "SWE_StarPU_Block.h"
#include <starpu/SWE_HUV_Matrix.h>
#include <hip/hip_runtime.h>
#include "codelets.h"
#include <cfloat>
#include <iostream>

#if defined(SOLVER_AUGRIE)

#include "solvers/AugRieCUDA.h"

__device__
void waveSolverCuda(float_type hLeft, float_type hRight,float huLeft, float huRight,float bLeft, float bRight,float& hUpdateLeft, float & hUpdateRight, float & huUpdateLeft, float& huUpdateRight, float & maxWaveSpeed){
    float results[5];
    augRieComputeNetUpdates(hLeft,hRight,
            huLeft,huRight,
            bLeft,bRight,
            SWE_StarPU_Block::g,static_cast<real>(0.01), static_cast<real>(0.000001), static_cast<real>(0.0001), 10,results);
    hUpdateLeft = results[0];
    hUpdateRight = results[1];
    huUpdateLeft = results[2];
    huUpdateRight = results[3];
    maxWaveSpeed = results[4];
}

#endif

#define CUDA_THREADS_PER_BLOCK 64

__device__ static float atomicMin(float *address, float val) {
    int *address_as_i = (int *) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

template<BoundaryEdge side>
__global__
void
computeNumericalFluxes_border(SWE_HUV_Matrix_interface mainBlock, SWE_HUV_Matrix_interface neighbourBlock,
                              starpu_matrix_interface b, float *maxTimestep, SWE_HUV_Matrix_interface netUpdates,
                              const uint32_t n,
                              float dX_inv, float dY_inv, float dX, float dY) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float maxEdgeSpeed[CUDA_THREADS_PER_BLOCK];
    if (index >= n) {
        return;
    }
    float_type hNetUpNeighbour, hNetUpMain;
    float_type huNetUpNeighbour, huNetUpMain;

    const uint32_t mainBlockX = side == BND_LEFT ? 0 : (side == BND_RIGHT ? mainBlock.nX - 1 : index);
    const uint32_t mainBlockY = side == BND_TOP ? 0 : (side == BND_BOTTOM ? mainBlock.nY - 1 : index);
    const uint32_t neighbourX = side == BND_LEFT || side == BND_RIGHT ? 0 : index;
    const uint32_t neighbourY = side == BND_TOP || BND_BOTTOM ? 0 : index;
    float_type hNeighbour = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&neighbourBlock, neighbourX, neighbourY);
    float_type hMain = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, mainBlockY, mainBlockX);
    float_type huNeighbour = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&neighbourBlock, neighbourX, neighbourY);
    float_type huMain = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&mainBlock, mainBlockY, mainBlockX);
    float_type bNeighbour = ((float_type * )
            STARPU_MATRIX_GET_PTR(&b))[
            (mainBlockY + (side == BND_BOTTOM ? 2 : (side == BND_TOP ? 0 : 1))) * STARPU_MATRIX_GET_LD(&b) +
            (mainBlockX + (side == BND_RIGHT ? 2 : (side == BND_LEFT ? 0 : 1)))];
    float_type bMain = ((float_type * )
            STARPU_MATRIX_GET_PTR(&b))[(mainBlockY + 1) * STARPU_MATRIX_GET_LD(&b) + (mainBlockX + 1)];


#if defined(SOLVER_AUGRIE)
    waveSolverCuda(
                        hNeighbour,  hMain,
                        huNeighbour, huMain,
                        bNeighbour,  bMain,
                        hNetUpNeighbour, hNetUpMain,
                        huNetUpNeighbour, huNetUpMain,
                        maxEdgeSpeed[threadIdx.x]
                       );
#endif


    if (side == BND_RIGHT || side == BND_LEFT) {
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, mainBlockY, mainBlockX) += dX_inv * hNetUpMain;
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, mainBlockY, mainBlockX) += dX_inv * huNetUpMain;
    } else {
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, mainBlockY, mainBlockX) += dY_inv * hNetUpMain;
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&netUpdates, mainBlockY, mainBlockX) += dY_inv * huNetUpMain;
    }
    __syncthreads();
    //Block wide reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            maxEdgeSpeed[threadIdx.x] = fmax(maxEdgeSpeed[threadIdx.x], maxEdgeSpeed[threadIdx.x + s]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        float localMaxTimeStep = FLT_MAX;
        if (maxEdgeSpeed[0] > 0) {
            localMaxTimeStep = fmin(dX / maxEdgeSpeed[0], dY / maxEdgeSpeed[0]) * SWECodelets::CFL_NUMBER;
        }
        atomicMin(maxTimestep, localMaxTimeStep);
    }

}

__global__
void computeNumericalFluxes_mainBlock(SWE_HUV_Matrix_interface mainBlock, starpu_matrix_interface b, float *maxTimestep,
                                      SWE_HUV_Matrix_interface netUpdates,
                                      float dX_inv, float dY_inv, float dX, float dY) {
    const auto idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const auto idxY = blockIdx.y * blockDim.y + threadIdx.y;

    const auto threadIdxLin = threadIdx.y*blockDim.x+threadIdx.x;

    __shared__ float maxEdgeSpeed[CUDA_THREADS_PER_BLOCK];

    if (idxX >= mainBlock.nX-1 || idxY >= mainBlock.nY-1) {
        return;
    }
    float_type hNetUpLeft, hNetUpRight;
    float_type huNetUpLeft, huNetUpRight;

    float_type hLeft = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY);
    float_type hRight = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX+1, idxY);
    float_type huLeft = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&mainBlock, idxX, idxY);
    float_type huRight = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&mainBlock, idxX+1, idxY);

    float_type bLeft = ((float_type*)(b.ptr))[(idxY+1)*b.ld+idxX+1];
    float_type bRight = ((float_type*)(b.ptr))[(idxY+1)*b.ld+idxX+2];

#if defined(SOLVER_AUGRIE)
    waveSolverCuda(
                        hLeft,  hRight,
                        huLeft, huRight,
                        bLeft, bRight,
                        hNetUpLeft, hNetUpRight,
                        huNetUpLeft, huNetUpRight,
                        maxEdgeSpeed[threadIdxLin]
                       );
#endif

    STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX, idxY) += dX_inv*hNetUpLeft;
    STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX+1, idxY) += dX_inv*hNetUpRight;
    STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, idxX, idxY) += dX_inv*huNetUpLeft;
    STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, idxX+1, idxY) += dX_inv*huNetUpRight;

    float_type hUpper = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY);
    float_type hLower = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY+1);
    float_type hvUpper = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY);
    float_type hvLower = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY+1);

    float_type bUpper = ((float_type*)(b.ptr))[(idxY+1)*b.ld+idxX+1];
    float_type bLower = ((float_type*)(b.ptr))[(idxY+2)*b.ld+idxX+1];
    float l_maxEdgeSpeed;
#if defined(SOLVER_AUGRIE)
    waveSolverCuda(
                        hUpper,  hLower,
                        hvUpper, hvLower,
                        bUpper, bLower,
                        hNetUpLeft, hNetUpRight,
                        huNetUpLeft, huNetUpRight,
                        l_maxEdgeSpeed
                       );
#endif
    maxEdgeSpeed[threadIdxLin] = fmax( maxEdgeSpeed[threadIdxLin],l_maxEdgeSpeed);

    STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY) += dY_inv*hNetUpLeft;
    STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY+1) += dY_inv*hNetUpRight;
    STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY)+= dY_inv*huNetUpLeft;
    STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY+1)+= dY_inv*huNetUpRight;

    __syncthreads();
    //Block wide reduction using shared memory
    for (unsigned int s = (blockDim.x*blockDim.y) / 2; s > 0; s >>= 1) {
        if (threadIdxLin < s) {
            maxEdgeSpeed[threadIdxLin] = fmax(maxEdgeSpeed[threadIdxLin], maxEdgeSpeed[threadIdxLin + s]);
        }
        __syncthreads();
    }
    if (threadIdxLin == 0) {
        float localMaxTimeStep = FLT_MAX;
        if (maxEdgeSpeed[0] > 0) {
            localMaxTimeStep = fmin(dX / maxEdgeSpeed[0], dY / maxEdgeSpeed[0]) * SWECodelets::CFL_NUMBER;
        }
        atomicMin(maxTimestep, localMaxTimeStep);
    }

}

void computeNumericalFluxes_cuda(void *buffers[], void *cl_arg) {
    const SWE_StarPU_Block *pBlock;
    starpu_codelet_unpack_args(cl_arg, &pBlock);

    const auto mainBlock = buffers[0];
    const auto leftGhost = buffers[1 + BND_LEFT];
    const auto rightGhost = buffers[1 + BND_RIGHT];
    const auto bottomGhost = buffers[1 + BND_BOTTOM];
    const auto topGhost = buffers[1 + BND_TOP];
    const auto b = buffers[5];
    const auto netUpdates = buffers[6];

    float *maxTimestep = (float *) STARPU_VARIABLE_GET_PTR(buffers[7]);

    const auto nX = pBlock->getNx();
    const auto nY = pBlock->getNy();

    const auto dX = pBlock->getDx();
    const auto dY = pBlock->getDy();

    const auto dX_inv = 1 / dX;
    const auto dY_inv = 1 / dY;

    const hipStream_t stream = starpu_cuda_get_local_stream();

    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_H_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);
    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_HU_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);
    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_HV_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);

    uint32_t gridSize = (nX + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    computeNumericalFluxes_border<BND_LEFT>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(leftGhost),
            *((starpu_matrix_interface *) (b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nY,
            dX_inv,
            dY_inv,
            dX,
            dY
    );
    computeNumericalFluxes_border<BND_RIGHT>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(rightGhost),
            *((starpu_matrix_interface *) (b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nY,
            dX_inv,
            dY_inv,
            dX,
            dY
    );

    computeNumericalFluxes_border<BND_TOP>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(topGhost),
            *((starpu_matrix_interface *) (b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nX,
            dX_inv,
            dY_inv,
            dX,
            dY
    );
    computeNumericalFluxes_border<BND_BOTTOM>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(bottomGhost),
            *((starpu_matrix_interface *) (b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nX,
            dX_inv,
            dY_inv,
            dX,
            dY
    );

    const auto blockWidth = std::floor(std::sqrt(CUDA_THREADS_PER_BLOCK));
    dim3 threads(blockWidth,blockWidth);
    dim3 blocks(
            (nX+threads.x-1)/threads.x,
            (nY+threads.y-1)/threads.y);
    computeNumericalFluxes_mainBlock<<<blocks,threads,0,stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            *((starpu_matrix_interface *) (b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            dX_inv,
            dY_inv,
            dX,
            dY
            );

    hipStreamSynchronize(stream);

}

__global__
void variableMin_cuda_kernel(float *a, float *b) {
    *a = std::min(*a, *b);
}

void variableMin_cuda(void *buffers[], void *cl_args) {
    float *a = (float *) STARPU_VARIABLE_GET_PTR(buffers[0]);
    float *b = (float *) STARPU_VARIABLE_GET_PTR(buffers[1]);
    const auto stream = starpu_cuda_get_local_stream();
    variableMin_cuda_kernel<<<1, 1, 0, stream>>>(a, b);
    hipStreamSynchronize(stream);
}

__global__
void variableSetInf_cuda_kernel(float *value) {
    *value = INFINITY;
}

void variableSetInf_cuda(void *buffers[], void *cl_args) {
    float *value = (float *) STARPU_VARIABLE_GET_PTR(buffers[0]);
    const auto stream = starpu_cuda_get_local_stream();
    variableSetInf_cuda_kernel<<<1, 1, 0, stream>>>(value);
    hipStreamSynchronize(stream);
}

