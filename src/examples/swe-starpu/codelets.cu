#include "hip/hip_runtime.h"
#include "codelets.cuh"
#include "SWE_StarPU_Block.h"
#include <starpu/SWE_HUV_Matrix.h>
#include <hip/hip_runtime.h>
#include "codelets.h"
#include <cfloat>
#include <iostream>

#if defined(SOLVER_AUGRIE)

#include "solvers/AugRieCUDA.h"

__device__
void waveSolverCuda(float_type hLeft, float_type hRight,float huLeft, float huRight,float bLeft, float bRight,float& hUpdateLeft, float & hUpdateRight, float & huUpdateLeft, float& huUpdateRight, float & maxWaveSpeed){
    float results[5];
    augRieComputeNetUpdates(hLeft,hRight,
            huLeft,huRight,
            bLeft,bRight,
            SWE_StarPU_Block::g,static_cast<real>(0.01), static_cast<real>(0.000001), static_cast<real>(0.0001), 10,results);
    hUpdateLeft = results[0];
    hUpdateRight = results[1];
    huUpdateLeft = results[2];
    huUpdateRight = results[3];
    maxWaveSpeed = results[4];
}

#endif

#define CUDA_THREADS_PER_BLOCK 64

__device__ static float atomicMin(float *address, float val) {
    int *address_as_i = (int *) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

template<BoundaryEdge side>
__global__
void
computeNumericalFluxes_border(SWE_HUV_Matrix_interface mainBlock, SWE_HUV_Matrix_interface neighbourBlock,
                              starpu_matrix_interface b, float *maxTimestep, SWE_HUV_Matrix_interface netUpdates,
                              const uint32_t n,
                              float dX_inv, float dY_inv, float dX, float dY) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float maxEdgeSpeed[CUDA_THREADS_PER_BLOCK];
    maxEdgeSpeed[threadIdx.x] = 0;
    if (index < n) {
        if (side == BND_RIGHT || side == BND_LEFT) {
            float_type hNetUpLeft, hNetUpRight;
            float_type huNetUpLeft, huNetUpRight;

            float_type hLeft = STARPU_SWE_HUV_MATRIX_GET_H_VAL(
                    side == BND_LEFT ? &neighbourBlock : &mainBlock,
                    side == BND_LEFT ? 0 : (mainBlock.nX - 1),
                    index);
            float_type hRight = STARPU_SWE_HUV_MATRIX_GET_H_VAL(
                    side == BND_LEFT ? &mainBlock : &neighbourBlock,
                    0,
                    index);
            float_type huLeft = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(
                    side == BND_LEFT ? &neighbourBlock : &mainBlock,
                    side == BND_LEFT ? 0 : (mainBlock.nX - 1),
                    index);
            float_type huRight = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(
                    side == BND_LEFT ? &mainBlock : &neighbourBlock,
                    0,
                    index);
            float_type bLeft = ((float_type *)
                    STARPU_MATRIX_GET_PTR(&b))[
                    (index + 1) * STARPU_MATRIX_GET_LD(&b) +
                    (side == BND_LEFT ? 0 : mainBlock.nX)];
            float_type bRight = ((float_type *)
                    STARPU_MATRIX_GET_PTR(&b))[(index + 1) * STARPU_MATRIX_GET_LD(&b) + 1 +
                                               (side == BND_LEFT ? 0 : mainBlock.nX)];


            waveSolverCuda(
                    hLeft, hRight,
                    huLeft, huRight,
                    bLeft, bRight,
                    hNetUpLeft, hNetUpRight,
                    huNetUpLeft, huNetUpRight,
                    maxEdgeSpeed[threadIdx.x]
            );

            if (side == BND_LEFT) {
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, 0, index) += dX_inv * hNetUpRight;
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, 0, index) += dX_inv * huNetUpRight;
            } else {
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, netUpdates.nX - 1, index) += dX_inv * hNetUpLeft;
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, netUpdates.nX - 1, index) += dX_inv * huNetUpLeft;
            }
        } else {
            float_type hNetUpUpper, hNetUpLower;
            float_type hvNetUpUpper, hvNetUpLower;

            float_type hUpper = STARPU_SWE_HUV_MATRIX_GET_H_VAL(
                    side == BND_TOP ? &neighbourBlock : &mainBlock,
                    index + (side == BND_TOP ? 1 : 0),
                    side == BND_TOP ? 0 : (mainBlock.nY - 1)
            );
            float_type hLower = STARPU_SWE_HUV_MATRIX_GET_H_VAL(
                    side == BND_TOP ? &mainBlock : &neighbourBlock,
                    side == BND_TOP ? index : (index + 1),
                    0
            );
            float_type hvUpper = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(
                    side == BND_TOP ? &neighbourBlock : &mainBlock,
                    index + (side == BND_TOP ? 1 : 0),
                    side == BND_TOP ? 0 : (mainBlock.nY - 1)
            );
            float_type hvLower = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(
                    side == BND_TOP ? &mainBlock : &neighbourBlock,
                    side == BND_TOP ? index : (index + 1),
                    0
            );

            float_type bUpper = ((float_type *)
                    STARPU_MATRIX_GET_PTR(&b))[(side == BND_TOP ? 0 : mainBlock.nY) *
                                               STARPU_MATRIX_GET_LD(&b) + (index + 1)];
            float_type bLower = ((float_type *)
                    STARPU_MATRIX_GET_PTR(&b))[((side == BND_TOP ? 0 : mainBlock.nY) + 1) *
                                               STARPU_MATRIX_GET_LD(&b) + (index + 1)];
            waveSolverCuda(hUpper, hLower,
                           hvUpper, hvLower,
                           bUpper, bLower,
                           hNetUpUpper, hNetUpLower,
                           hvNetUpUpper, hvNetUpLower,
                           maxEdgeSpeed[threadIdx.x]);
            if (side == BND_TOP) {
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, index, 0) += dY_inv * hNetUpLower;
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&netUpdates, index, 0) += dY_inv * hvNetUpLower;
            } else {
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, index, mainBlock.nY - 1) += dY_inv * hNetUpUpper;
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&netUpdates, index, mainBlock.nY - 1) += dY_inv * hvNetUpUpper;
            }
        }
    }

    __syncthreads();
//Block wide reduction using shared memory
    for (unsigned int s = blockDim.x / 2;s > 0; s >>= 1) {
        if (threadIdx.x < s && threadIdx.x + s < CUDA_THREADS_PER_BLOCK) {
            maxEdgeSpeed[threadIdx.x] = fmax(maxEdgeSpeed[threadIdx.x], maxEdgeSpeed[threadIdx.x + s]);
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        float localMaxTimeStep = FLT_MAX;
        if (maxEdgeSpeed[0] > 0) {
            localMaxTimeStep = fmin(dX / maxEdgeSpeed[0], dY / maxEdgeSpeed[0]) * SWECodelets::CFL_NUMBER;
        }
        atomicMin(maxTimestep, localMaxTimeStep);
    }
}

__global__
void computeNumericalFluxes_mainBlock(SWE_HUV_Matrix_interface mainBlock, starpu_matrix_interface b, float *maxTimestep,
                                      SWE_HUV_Matrix_interface netUpdates,
                                      float dX_inv, float dY_inv, float dX, float dY) {
    const auto idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const auto idxY = blockIdx.y * blockDim.y + threadIdx.y;

    const auto threadIdxLin = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ float maxEdgeSpeed[CUDA_THREADS_PER_BLOCK];
    maxEdgeSpeed[threadIdxLin] = 0;
    if (idxX > 0 && idxX < mainBlock.nX && idxY < mainBlock.nY) {
        float_type hNetUpLeft, hNetUpRight;
        float_type huNetUpLeft, huNetUpRight;

        float_type hLeft = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX - 1, idxY);
        float_type hRight = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY);
        float_type huLeft = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&mainBlock, idxX - 1, idxY);
        float_type huRight = STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&mainBlock, idxX, idxY);

        float_type bLeft = ((float_type * )(b.ptr))[(idxY + 1) * b.ld + idxX];
        float_type bRight = ((float_type * )(b.ptr))[(idxY + 1) * b.ld + idxX + 1];
        float l_maxEdgeSpeed;
        waveSolverCuda(
                hLeft, hRight,
                huLeft, huRight,
                bLeft, bRight,
                hNetUpLeft, hNetUpRight,
                huNetUpLeft, huNetUpRight,
                l_maxEdgeSpeed
        );
        maxEdgeSpeed[threadIdxLin] = fmax(maxEdgeSpeed[threadIdxLin], l_maxEdgeSpeed);
        /*if(hNetUpLeft!= 0 || huNetUpLeft != 0 || hNetUpRight != 0 || huNetUpRight != 0) {
            printf("netUpLeft: %f, netUpRight: %f, huNetUpLeft: %f, huNetUpRight: %f\n", hNetUpLeft, hNetUpRight,
                   huNetUpLeft, huNetUpRight);
        }*/
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX - 1, idxY) += dX_inv * hNetUpLeft;
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX, idxY) += dX_inv * hNetUpRight;
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, idxX - 1, idxY) += dX_inv * huNetUpLeft;
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&netUpdates, idxX, idxY) += dX_inv * huNetUpRight;
    }
    if (idxX < mainBlock.nX && idxY < mainBlock.nY-1) {
        float_type hUpper = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY);
        float_type hLower = STARPU_SWE_HUV_MATRIX_GET_H_VAL(&mainBlock, idxX, idxY + 1);
        float_type hvUpper = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY);
        float_type hvLower = STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&mainBlock, idxX, idxY + 1);

        float_type bUpper = ((float_type * )(b.ptr))[(idxY + 1) * b.ld + idxX + 1];
        float_type bLower = ((float_type * )(b.ptr))[(idxY + 2) * b.ld + idxX + 1];

        float_type hNetUpUpper, hNetUpLower;
        float_type hvNetUpUpper, hvNetUpLower;
        float l_maxEdgeSpeed;
        waveSolverCuda(
                hUpper, hLower,
                hvUpper, hvLower,
                bUpper, bLower,
                hNetUpUpper, hNetUpLower,
                hvNetUpUpper, hvNetUpLower,
                l_maxEdgeSpeed
        );

        maxEdgeSpeed[threadIdxLin] = fmax(maxEdgeSpeed[threadIdxLin], l_maxEdgeSpeed);

        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX, idxY) += dY_inv * hNetUpUpper;
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&netUpdates, idxX, idxY + 1) += dY_inv * hNetUpLower;
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&netUpdates, idxX, idxY) += dY_inv * hvNetUpUpper;
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&netUpdates, idxX, idxY + 1) += dY_inv * hvNetUpLower;
    }
    __syncthreads();
    //Block wide reduction using shared memory
    for (unsigned int s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1) {
        if (threadIdxLin < s) {
            maxEdgeSpeed[threadIdxLin] = fmax(maxEdgeSpeed[threadIdxLin], maxEdgeSpeed[threadIdxLin + s]);
        }
        __syncthreads();
    }
    if (threadIdxLin == 0) {
        float localMaxTimeStep = FLT_MAX;
        if (maxEdgeSpeed[0] > 0) {
            localMaxTimeStep = fmin(dX / maxEdgeSpeed[0], dY / maxEdgeSpeed[0]) * SWECodelets::CFL_NUMBER;
        }
        atomicMin(maxTimestep, localMaxTimeStep);
    }

}

void computeNumericalFluxes_cuda(void *buffers[], void *cl_arg) {
    const SWE_StarPU_Block *pBlock;
    starpu_codelet_unpack_args(cl_arg, &pBlock);

    const auto mainBlock = buffers[0];
    const auto leftGhost = buffers[1 + BND_LEFT];
    const auto rightGhost = buffers[1 + BND_RIGHT];
    const auto bottomGhost = buffers[1 + BND_BOTTOM];
    const auto topGhost = buffers[1 + BND_TOP];
    const auto b = buffers[5];
    const auto netUpdates = buffers[6];

    float *maxTimestep = (float *) STARPU_VARIABLE_GET_PTR(buffers[7]);

    const auto nX = pBlock->getNx();
    const auto nY = pBlock->getNy();

    const auto dX = pBlock->getDx();
    const auto dY = pBlock->getDy();

    const auto dX_inv = 1 / dX;
    const auto dY_inv = 1 / dY;

    const hipStream_t stream = starpu_cuda_get_local_stream();

    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_H_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);
    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_HU_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);
    hipMemsetAsync(STARPU_SWE_HUV_MATRIX_GET_HV_PTR(netUpdates), 0,
                    sizeof(float_type) * nX * nY, stream);

    uint32_t gridSize = (nX + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    computeNumericalFluxes_border<BND_LEFT>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(leftGhost),
            *((starpu_matrix_interface * )(b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nY,
            dX_inv,
            dY_inv,
            dX,
            dY
    );
    computeNumericalFluxes_border<BND_RIGHT>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(rightGhost),
            *((starpu_matrix_interface * )(b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nY,
            dX_inv,
            dY_inv,
            dX,
            dY
    );

    computeNumericalFluxes_border<BND_TOP>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(topGhost),
            *((starpu_matrix_interface * )(b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nX,
            dX_inv,
            dY_inv,
            dX,
            dY
    );
    computeNumericalFluxes_border<BND_BOTTOM>
    <<<gridSize, CUDA_THREADS_PER_BLOCK, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(bottomGhost),
            *((starpu_matrix_interface * )(b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            nX,
            dX_inv,
            dY_inv,
            dX,
            dY
    );

    const auto blockWidth = std::floor(std::sqrt(CUDA_THREADS_PER_BLOCK));
    dim3 threads(blockWidth, blockWidth);
    dim3 blocks(
            (nX + threads.x - 1) / threads.x,
            (nY + threads.y - 1) / threads.y);
    computeNumericalFluxes_mainBlock<<<blocks, threads, 0, stream>>>(
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(mainBlock),
            *((starpu_matrix_interface * )(b)),
            maxTimestep,
            STARPU_SWE_HUV_MATRIX_GET_INTERFACE(netUpdates),
            dX_inv,
            dY_inv,
            dX,
            dY
    );

    hipStreamSynchronize(stream);

}

__global__
void variableMin_cuda_kernel(float *a, float *b) {
    *a = std::min(*a, *b);
}

void variableMin_cuda(void *buffers[], void *cl_args) {
    float *a = (float *) STARPU_VARIABLE_GET_PTR(buffers[0]);
    float *b = (float *) STARPU_VARIABLE_GET_PTR(buffers[1]);
    const auto stream = starpu_cuda_get_local_stream();
    variableMin_cuda_kernel<<<1, 1, 0, stream>>>(a, b);
    hipStreamSynchronize(stream);
}

__global__
void variableSetInf_cuda_kernel(float *value) {
    *value = INFINITY;
}

void variableSetInf_cuda(void *buffers[], void *cl_args) {
    float *value = (float *) STARPU_VARIABLE_GET_PTR(buffers[0]);
    const auto stream = starpu_cuda_get_local_stream();
    variableSetInf_cuda_kernel<<<1, 1, 0, stream>>>(value);
    hipStreamSynchronize(stream);
}

__global__
void updateUnkowns_cuda_kernel(const SWE_HUV_Matrix_interface myBlock, const SWE_HUV_Matrix_interface updates,
                               const float *const dt,
                               const size_t nX,
                               const size_t nY) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= nX || y >= nY) {
        return;
    }
    STARPU_SWE_HUV_MATRIX_GET_H_VAL(&myBlock, x, y) -= *dt * STARPU_SWE_HUV_MATRIX_GET_H_VAL(&updates, x, y);
    STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&myBlock, x, y) -= *dt * STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&updates, x, y);
    STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&myBlock, x, y) -= *dt * STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&updates, x, y);
    if (STARPU_SWE_HUV_MATRIX_GET_H_VAL(&myBlock, x, y) < SWECodelets::DRY_LIMIT) {
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(&myBlock, x, y) =
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(&myBlock, x, y) =
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(&myBlock, x, y) = 0;
    }
}

void updateUnknowns_cuda(void *buffers[], void *cl_args) {
    const SWE_StarPU_Block *pBlock;
    starpu_codelet_unpack_args(cl_args, &pBlock);

    const auto myBlock = buffers[0];
    const auto updates = buffers[1];
    const auto dt = (const float *) STARPU_VARIABLE_GET_PTR(buffers[2]);

    const hipStream_t stream = starpu_cuda_get_local_stream();

    const auto blockWidth = std::floor(std::sqrt(CUDA_THREADS_PER_BLOCK));
    dim3 threads(blockWidth, blockWidth);
    dim3 blocks(
            (pBlock->getNx() + threads.x - 1) / threads.x,
            (pBlock->getNy() + threads.y - 1) / threads.y);

    updateUnkowns_cuda_kernel<<<blocks, threads, 0, stream>>>(STARPU_SWE_HUV_MATRIX_GET_INTERFACE(myBlock),
                                                              STARPU_SWE_HUV_MATRIX_GET_INTERFACE(updates),
                                                              dt,
                                                              pBlock->getNx(),
                                                              pBlock->getNy());

    hipStreamSynchronize(stream);
}

void updateGhostLayers_cuda(void *buffers[], void *cl_arg) {
    const SWE_StarPU_Block *thisBlock;
    BoundaryEdge side;
    starpu_codelet_unpack_args(cl_arg, &side, &thisBlock);
#ifdef DBG
    cout << "Set simple boundary conditions " << endl << flush;
#endif
    auto myBlockData = buffers[1];
    auto myBorderData = buffers[0];


    const bool vertical = side == BND_LEFT || side == BND_RIGHT;
    const auto nx = STARPU_SWE_HUV_MATRIX_GET_NX(myBlockData);
    const auto ny = STARPU_SWE_HUV_MATRIX_GET_NY(myBlockData);

    switch (thisBlock->boundary[side]) {
        case WALL:
        case OUTFLOW: {
            const bool wall = thisBlock->boundary[side] == WALL;
#ifdef VECTORIZE
#pragma omp simd
#endif
            for (size_t j = 0; j < (vertical ? ny : nx); j++) {
                const size_t outerX = vertical ? 0 : j + 1;
                const size_t innerX = side == BND_LEFT ? 0 : (side == BND_RIGHT ? nx - 1 : j);
                const size_t outerY = vertical ? j : 0;
                const size_t innerY = side == BND_TOP ? 0 : (side == BND_BOTTOM ? ny - 1 : j);

                STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, outerX, outerY) =
                        STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBlockData, innerX, innerY);
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, outerX, outerY) = (vertical && wall ? -1.f : 1.f) *
                                                                                 STARPU_SWE_HUV_MATRIX_GET_HU_VAL(
                                                                                         myBlockData, innerX, innerY);
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, outerX, outerY) = (!vertical && wall ? -1.f : 1.f) *
                                                                                 STARPU_SWE_HUV_MATRIX_GET_HV_VAL(
                                                                                         myBlockData, innerX, innerY);

            }
        }
            break;
        case CONNECT:
        case PASSIVE:
            break;
        default:
            assert(false);
            break;
    }

    if (thisBlock->boundary[side] == CONNECT) {
        auto myNeighbourData = buffers[2];
        const auto neighbourNX = STARPU_SWE_HUV_MATRIX_GET_NX(myNeighbourData);
        const auto neighbourNY = STARPU_SWE_HUV_MATRIX_GET_NY(myNeighbourData);
#ifdef VECTORIZE
#pragma omp simd
#endif
        for (size_t i = 0; i < (vertical ? ny : nx); ++i) {
            const size_t boundaryX = vertical ? 0 : 1 + i;
            const size_t boundaryY = vertical ? i : 0;

            const size_t neighbourX = side == BND_LEFT ? neighbourNX - 1 : (side == BND_RIGHT ? 0 : i);
            const size_t neigbhourY = side == BND_TOP ? neighbourNY - 1 : (side == BND_BOTTOM ? 0 : i);

            STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, boundaryX, boundaryY) =
                    STARPU_SWE_HUV_MATRIX_GET_H_VAL(myNeighbourData, neighbourX, neigbhourY);
            STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, boundaryX, boundaryY) =
                    STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myNeighbourData, neighbourX, neigbhourY);
            STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, boundaryX, boundaryY) =
                    STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myNeighbourData, neighbourX, neigbhourY);
        }
    }
    //Update the corner values only the top and bottom boundary contain these
    if (side == BND_TOP) {
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBlockData, 0, 0);
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBlockData, 0, 0);
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBlockData, 0, 0);

        STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBlockData, nx - 1, 0);
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBlockData, nx - 1, 0);
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBlockData, nx - 1, 0);
    }
    if (side == BND_BOTTOM) {
        STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBlockData, 0, ny - 1);
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBlockData, 0, ny - 1);
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, 0, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBlockData, 0, ny - 1);

        STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_H_VAL(myBlockData, nx - 1, ny - 1);
        STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HU_VAL(myBlockData, nx - 1, ny - 1);
        STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBorderData, nx + 1, 0) =
                STARPU_SWE_HUV_MATRIX_GET_HV_VAL(myBlockData, nx - 1, ny - 1);
    }

#ifdef DBG
    cout << "Set CONNECT boundary conditions in main memory " << endl << flush;
#endif


}