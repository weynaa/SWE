#include "hip/hip_runtime.h"
#include "testkernels.cuh"
#include <cstdlib>
#define STARPU_USE_CUDA
#include <starpu.h>

__global__ void  test_cuda_kernel(float *val, float factor, size_t nx, size_t ny, size_t stride) {
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < nx && y < ny) {
        val[y*stride+x] *= factor;
    }
}

void test_cuda_func(void *buffers[], void *args) {
    float *factor = (float *) args;
    /* length of the vector */
    unsigned nx = STARPU_MATRIX_GET_NX(buffers[0]);
    unsigned ny = STARPU_MATRIX_GET_NY(buffers[0]);
    unsigned row_stride = STARPU_MATRIX_GET_LD(buffers[0]);
    /* local copy of the vector pointer */
    float *val = (float *) STARPU_MATRIX_GET_PTR(buffers[0]);
    dim3 threads_per_block = {8,8};
    dim3 nblocks = {(nx + threads_per_block.x - 1) / threads_per_block.x,
                    (ny + threads_per_block.y - 1) / threads_per_block.y};
    test_cuda_kernel<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>(val, *factor, nx,ny,row_stride);
    hipStreamSynchronize(starpu_cuda_get_local_stream());
}