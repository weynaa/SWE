#include "hip/hip_runtime.h"
/**
 * @file
 * This file is part of SWE.
 *
 * @author Alexander Breuer (breuera AT in.tum.de, http://www5.in.tum.de/wiki/index.php/Dipl.-Math._Alexander_Breuer)
 *         Michael Bader (bader AT in.tum.de, http://www5.in.tum.de/wiki/index.php/Univ.-Prof._Dr._Michael_Bader)
 *
 * @section LICENSE
 *
 * SWE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SWE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SWE.  If not, see <http://www.gnu.org/licenses/>.
 *
 *
 * @section DESCRIPTION
 *
 * Basic setting of SWE, which uses a wave propagation solver and an artificial or ASAGI scenario on a single block.
 */

#include <cassert>
#include <cstdlib>
#include <string>
#include <iostream>

#include "blocks/SWE_Block.hh"

#include "writer/Writer.hh"

#ifdef ASAGI
#include "scenarios/SWE_AsagiScenario.hh"
#else

#include "scenarios/SWE_simple_scenarios.hh"

#endif

#ifdef READXML
#include "tools/CXMLConfig.hpp"
#endif

#include "tools/args.hh"
#include "tools/help.hh"
#include "tools/Logger.hh"
#include "tools/ProgressBar.hh"

#define STARPU_USE_CUDA
#define STARPU_USE_MPI
#include <starpu.h>

static void test_cpu_func(void *buffers[], void *_args) {
    float *factor = (float *) _args;
    /* length of the vector */
    unsigned nx = STARPU_MATRIX_GET_NX(buffers[0]);
    unsigned ny = STARPU_MATRIX_GET_NY(buffers[0]);
    /* local copy of the vector pointer */
    unsigned row_stride = STARPU_MATRIX_GET_LD(buffers[0]);
    float *val = (float *) STARPU_MATRIX_GET_PTR(buffers[0]);

    for (auto y = 0; y < ny; ++y) {
        for(auto x = 0; x < nx;++x) {
            val[y*row_stride+x] *= *factor;
        }
    }
}


static void __global__ test_cuda_kernel(float *val, float factor, size_t nx, size_t ny, size_t stride) {
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < nx && y < ny) {
        val[y*stride+x] *= factor;
    }
}

static void test_cuda_func(void *buffers[], void *args) {
    float *factor = (float *) args;
    /* length of the vector */
    unsigned nx = STARPU_MATRIX_GET_NX(buffers[0]);
    unsigned ny = STARPU_MATRIX_GET_NY(buffers[0]);
    unsigned offset = STARPU_MATRIX_GET_OFFSET(buffers[0]);
    unsigned row_stride = STARPU_MATRIX_GET_LD(buffers[0]);
    const auto n = nx*ny;
    /* local copy of the vector pointer */
    float *val = (float *) STARPU_MATRIX_GET_PTR(buffers[0]);
    dim3 threads_per_block = {8,8};
    dim3 nblocks = {(nx + threads_per_block.x - 1) / threads_per_block.x,
                    (ny + threads_per_block.y - 1) / threads_per_block.y};
    test_cuda_kernel<<<nblocks, threads_per_block, 0, starpu_cuda_get_local_stream()>>>(val, *factor, nx,ny,row_stride);
    hipStreamSynchronize(starpu_cuda_get_local_stream());
}

static starpu_codelet test_codelet = []() {
    //Only C++20 has designated initializers
    starpu_codelet codelet = {};
    codelet.where = STARPU_CPU | STARPU_CUDA;
    codelet.nbuffers = 1;
    codelet.cpu_funcs[0] = test_cpu_func;
    codelet.modes[0] = STARPU_RW;
    codelet.cuda_funcs[0] = test_cuda_func;
    return codelet;
}();


/**
 * Main program for the simulation on a single SWE_WavePropagationBlock.
 */
int main(int argc, char **argv) {
    /**
     * Initialization.
     */
    // Parse command line parameters
    tools::Args args;
    args.addOption("grid-size-x", 'x', "Number of cells in x direction");
    args.addOption("grid-size-y", 'y', "Number of cells in y direction");
    args.addOption("output-basepath", 'o', "Output base file name");

    tools::Args::Result ret = args.parse(argc, argv);

    switch (ret) {
        case tools::Args::Error:
            return 1;
        case tools::Args::Help:
            return 0;
        default:
            break;
    }

    //! number of grid cells in x- and y-direction.
    int l_nX, l_nY;


    //! l_baseName of the plots.
    std::string l_baseName;

    // read command line parameters
    l_nX = args.getArgument<int>("grid-size-x");
    l_nY = args.getArgument<int>("grid-size-y");
    l_baseName = args.getArgument<std::string>("output-basepath");

    starpu_conf conf = {};
    starpu_conf_init(&conf);
    //conf.ncuda=0;
    //conf.nopencl=0;
    auto starpuret = starpu_init(&conf);
    if (starpuret != 0) {
        std::cerr << "Could not initialize StarPU!\n";
        return 1;
    }

    printf("StarPU workers:\n");
    printf("%d CPU cores\n", starpu_worker_get_count_by_type(STARPU_CPU_WORKER));
    printf("%d CUDA GPUs\n", starpu_worker_get_count_by_type(STARPU_CUDA_WORKER));
    printf("%d OpenCL GPUs\n", starpu_worker_get_count_by_type(STARPU_OPENCL_WORKER));

    const auto grid_Data = (float *) malloc(sizeof(float) * l_nX * l_nY);
    for (int i = 0; i < l_nX*l_nY; ++i) {
        grid_Data[i] = 1.0f;
    }
    starpu_data_handle_t dataHandle;
    starpu_matrix_data_register(&dataHandle, STARPU_MAIN_RAM, (uintptr_t) grid_Data, l_nX, l_nX, l_nY,
                                sizeof(grid_Data[0]));
    const float factor = 2.12116315;

    constexpr int NTASKS = 10;
    starpu_data_filter dataFilter = {};
    dataFilter.filter_func = starpu_matrix_filter_block;
    dataFilter.nchildren = NTASKS;
    starpu_data_partition(dataHandle, &dataFilter);

    for(int i = 0; i < starpu_data_get_nb_children(dataHandle);++i)
    {
        starpu_data_handle_t partHandle = starpu_data_get_sub_data(dataHandle,1,i);
        auto task = starpu_task_create();

        task->synchronous = 1;
        task->cl = &test_codelet;
        task->handles[0] = partHandle;
        task->cl_arg = (void *) &factor;
        task->cl_arg_size = sizeof(factor);

        starpu_task_submit(task);
    }
    for(int i = 0; i < starpu_data_get_nb_children(dataHandle);++i){
        starpu_data_unpartition(dataHandle,i);
    }

    starpu_data_unregister(dataHandle);

    for (int i = 0; i < l_nX*l_nY; ++i) {
        if(grid_Data[i] != factor)
        {
            printf("error");
        }
    }
    starpu_shutdown();
    return 0;
}
